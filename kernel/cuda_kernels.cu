
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void derifact_multiplication(double *matr, double *mati, double *vecr, double *veci, int fftlen, int fftnum)
{
    int index_x = blockIdx.x*blockDim.x + threadIdx.x; 
    int index_y = blockIdx.y*blockDim.y + threadIdx.y;

    int matindex = index_y*fftlen+index_x; //mat should be a contiguous array
    //printf("Block(x,y): (%d,%d). Thread(x,y): (%d,%d)\\n",blockIdx.x,blockIdx.y,threadIdx.x,threadIdx.y);
    // if N1%16>0, we're starting too many threads.
    // There is probably a better way to do this, but just eating the surplus should work.
    if (matindex < fftlen*fftnum) {
        double matreal = matr[matindex];
        double matimag = mati[matindex];
        double vecreal = vecr[index_x];
        double vecimag = veci[index_x];

        matr[matindex] = matreal*vecreal - matimag*vecimag;
        mati[matindex] = matreal*vecimag + matimag*vecreal;
    }
}

__global__ void pressure_window_multiplication(double *mr, double *mi, double *A, double *p1, double *p2, double *p3, int winlen, int Ns1, int Ns2, int Ns3, int fftlen, int fftnum, double R21, double R00, double R31, double R10) //passing a few by value seems to be more efficient than building an array first in pycuda
{
    int index_x = blockIdx.x*blockDim.x + threadIdx.x; 
    int index_y = blockIdx.y*blockDim.y + threadIdx.y;

    if (index_y < fftnum) { //eat the surplus
        int matindex = index_y*fftlen+index_x;

        double G = 1;
        if (index_x < winlen) {
            G = A[index_x];
        } else if (index_x > winlen+Ns2-1 && index_x < winlen*2+Ns2) {
            G = A[index_x-Ns2];
        }
        mi[matindex] = 0;
        if (index_x < winlen) {
            mr[matindex] = G*(R21*p1[Ns1*index_y+index_x-winlen+Ns1] + R00*p2[Ns2*index_y+winlen-1-index_x]);
        } else if (index_x < winlen + Ns2) {
            mr[matindex] = p2[Ns2*index_y+index_x-winlen];
        } else if (index_x < winlen*2+Ns2) {
            mr[matindex] = G*(R31*p3[Ns3*index_y+index_x-winlen-Ns2] + R10*p2[Ns2*index_y+2*Ns2+winlen-1-index_x]);
        } else {
            mr[matindex] = 0; //zero padding
        }
        //if(mr[matindex]==0 && matindex < 50) printf("zero at:%d\\n",matindex%fftlen);
    }
}

__global__ void velocity_window_multiplication(double *mr, double *mi, double *A, double *p1, double *p2, double *p3, int winlen, int Ns1, int Ns2, int Ns3, int fftlen, int fftnum, double R21, double R00, double R31, double R10) //passing a few by value seems to be more efficient than building an array first in pycuda
{
    int index_x = blockIdx.x*blockDim.x + threadIdx.x; 
    int index_y = blockIdx.y*blockDim.y + threadIdx.y;

    if (index_y < fftnum) { //eat the surplus
        int matindex = index_y*fftlen+index_x;

        double G = 1;
        if (index_x < winlen) {
            G = A[index_x];
        } else if (index_x > winlen+Ns2-1 && index_x < winlen*2+Ns2) {
            G = A[index_x-Ns2];
        }
        mi[matindex] = 0;
        if (index_x < winlen) {
            mr[matindex] = G*(R21*p1[Ns1*index_y+index_x-winlen+Ns1-1] + R00*p2[Ns2*index_y+winlen-index_x]);
        } else if (index_x < winlen + Ns2) {
            mr[matindex] = p2[Ns2*index_y+index_x-winlen];
        } else if (index_x < winlen*2+Ns2) {
            mr[matindex] = G*(R31*p3[Ns3*index_y+index_x-winlen-Ns2+1] + R10*p2[Ns2*index_y+2*Ns2+winlen-2-index_x]);
        } else {
            mr[matindex] = 0; //zero padding
        }
        //if(mr[matindex]==0 && matindex < 50) printf("zero at:%d\\n",matindex%fftlen);
    }
}
